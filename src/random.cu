#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "random.cuh"

__global__ void setup_radome(hiprandStateXORWOW_t *state, unsigned long long seed, unsigned long long N)
{
    unsigned long long id = threadIdx.x+blockDim.x*blockIdx.x ;
    if(id<N){
      /* Each thread gets same seed, a different sequence
         number, no offset */
      hiprand_init(seed, id, 0, &state[id]);
    }
}

__global__ void setup_radome(hiprandStatePhilox4_32_10_t *state, unsigned long long seed, unsigned long long N)
{
    unsigned long long id = threadIdx.x+blockDim.x*blockIdx.x;
    if(id<N){
      /* Each thread gets same seed, a different sequence
      number, no offset */
      hiprand_init(seed, id, 0, &state[id]);
    }
}

__global__ void generate_kernel(hiprandStatePhilox4_32_10_t *random,unsigned int *buffer,unsigned long long N, int t){
  unsigned long long id = threadIdx.x+blockDim.x*blockIdx.x;
  if(id<N){
    hiprandStatePhilox4_32_10_t localState = random[id];
    unsigned long long tid=t*id;
    for(int i=0;i<t;i++){
      buffer[tid+i]=hiprand(&localState);
    }
    random[id] = localState;
  }
}
